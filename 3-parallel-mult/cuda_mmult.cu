#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdbool.h>

double** create_matrix(int, int);
void matrix_init(double**, int, int);
__global__ void cuda_mmult(double* a, double* b, double* c, int size);
static inline long long timestamp();

int main(int argc, char* argv[]) {
    int size = 1600;
    int BLOCK_SIZE = 16;
    float tmmult;
    float nflop = 2.0 * (double)size * (double)size * (double)size;

    double** A0, ** B0, ** C0;

    A0 = create_matrix(size, size);
    B0 = create_matrix(size, size);
    C0 = create_matrix(size, size);

    if (!A0 || !B0 || !C0) {
        perror("Malloc Failed");
        exit(EXIT_FAILURE);
    }
    printf("Malloc completed\n");

    double* A = A0[0];
    double* B = B0[0];
    double* C = C0[0];

    matrix_init(A0, size, size, false);
    matrix_init(B0, size, size, false);
    matrix_init(C0, size, size, true);

    printf("Init completed\n");

    double* cuda_A = 0;
    double* cuda_B = 0;
    double* cuda_C = 0;

    hipMalloc(&cuda_A, size * size * sizeof(double));
    hipMalloc(&cuda_B, size * size * sizeof(double));
    hipMalloc(&cuda_C, size * size * sizeof(double));

    printf("hipMalloc completed\n");

    hipMemcpy(cuda_A, A, size * size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cuda_B, B, size * size * sizeof(double), hipMemcpyHostToDevice);

    printf("hipMemcpy completed\n");


    /* kernel invocation parameters */
    int  nblocks_x = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int  nblocks_y = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(nblocks_x, nblocks_y);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    /* START TEST*/

    hipEventRecord(start);
    cuda_mmult <<<dimGrid, dimBlock>>> (cuda_A, cuda_B, cuda_C, size);
    hipEventRecord(stop);
    /* END TEST*/

    hipMemcpy(C, cuda_C, size * size * sizeof(double), hipMemcpyDeviceToHost);

    double sum = 0.0;
    for (int i = 0; i < size && i < size; i++) {
        sum += C0[i][i];
    }

    hipEventElapsedTime(&tmmult, start, stop);
       
    float tmmult_s = tmmult / 1000.0;

    printf("matrix_size, gflops_mmult, tmmult, trace_mmult\n");
    printf("%d, %f, %f, %12.12g\n", size, nflop / (tmmult_s * 1.0e9), tmmult_s, sum);

    hipFree(cuda_A);
    hipFree(cuda_B);
    hipFree(cuda_C);

    free((void*)A0[0]);
    free((void*)B0[0]);
    free((void*)C0[0]);
    free((void*)A0);
    free((void*)B0);
    free((void*)C0);
}


double** create_matrix(int rows, int cols) {
    double** row_ptrs = (double**)malloc(rows * sizeof(double*));
    row_ptrs[0] = (double*)malloc(rows * cols * sizeof(double));

    for (int i = 1; i < rows; i++) {
        row_ptrs[i] = row_ptrs[0] + i * cols;
    }
    return row_ptrs;
}


void matrix_init(double** matrix, int rows, int cols, bool zeroes) {
    if(zeroes) {
        for (int i = 0; i < rows; i++) {
            for (int j = 0; j < cols; j++) {
                matrix[i][j] = 0.0;
            }
        }
    } else {
        for (int i = 0; i < rows; i++) {
            for (int j = 0; j < cols; j++) {
                matrix[i][j] = (double)(i) + (double)(j);
            }
        }
    }
}

__global__ void cuda_mmult(double* a, double* b, double* c, int size) {
    /* index calculation */
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    /* CUDA matrix multiplication */
    if (i < size && j < size) {
        for (int k = 0; k < size; ++k) {
            c[i * size + j] += a[i * size + k] * b[k * size + j];
        }
    }
}